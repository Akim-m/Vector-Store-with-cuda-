#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

extern "C" void calculate_dot_product(float* a, float* b, float* result, int n) {
    float *d_a, *d_b;
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    hipblasSdot(handle, n, d_a, 1, d_b, 1, result);

    hipFree(d_a);
    hipFree(d_b);

    hipblasDestroy(handle);
}

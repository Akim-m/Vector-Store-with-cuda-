#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" {

    float cuda_norm_cublas(float* h_vector, int n) {
        float norm_result = 0.0f;

        hipblasHandle_t handle;
        hipblasCreate(&handle);

        float* d_vector;
        hipMalloc(&d_vector, n * sizeof(float));

        hipblasSetVector(n, sizeof(float), h_vector, 1, d_vector, 1);

        hipblasSnrm2(handle, n, d_vector, 1, &norm_result);

        hipFree(d_vector);

        hipblasDestroy(handle);

        return norm_result;
    }
}
